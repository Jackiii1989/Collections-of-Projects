#include "hip/hip_runtime.h"

#include "device_module.h"
//#include "host_device_functions.h"
#include<math.h>
#include "Macros.h"



__device__  float MapToImaginary1(int y, float imax, float imin) {
	float range = imax - imin;
	return y * (range / HEIGHT) + imin;
}

#define LIMIT 4.0
__device__  int findMandelBrot1(float cr, float ci) {

	size_t i = 0;
	float zr = 0.0, zi = 0.0;

#if LOOPUNROLLING
#pragma unroll (MAXITER)
	for (; i < MAXITER && zr*zr + zi * zi < LIMIT; i++) {
		//if (zr*zr + zi * zi < LIMIT) { break; }
		float temp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		zi = 2.0f * zr * zi + ci;
		zr = temp; //now it is okay
		
	}
#else
	while (zr*zr + zi * zi < LIMIT && i < MAXITER) {
		float temp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		zi = 2.0f * zr * zi + ci;
		zr = temp; //now it is okay
		//if (zr*zr + zi * zi < LIMIT) { break; }
		i++;
	}
#endif
	return static_cast<int>(i);
}


//__global__  void call_kernel(pfc::BGR_4_t * bmp, ImageMetaData* md) {
__global__  void call_kernel(pix_t * bmp, float zoomLevel) {

	auto const x{ blockIdx.x * blockDim.x + threadIdx.x };
	auto const y{ blockIdx.y * blockDim.y + threadIdx.y };



	
	//pfc::byte_t k; // Iteration counter 
	//ci = MapToImaginary1(y, IMAX, IMIN); // map y to the imaginary parf of value
	//cr = MapToReal1(x, RMAX, RMIN); // map x to real part of the value
#if  SEC_APPROACH
#if INTRISIC
	float rmax = RCENTER - (__fdividef(RMIN, zoomLevel));
	float rmin = RCENTER + (__fdividef(RMIN, zoomLevel));
	float imax = ICENTER - (__fdividef(RMIN, zoomLevel));
	float imin = ICENTER + (__fdividef(RMIN, zoomLevel));
#else
	
	float rmax = RCENTER - (RMIN / zoomLevel);
	float rmin = RCENTER + (RMIN / zoomLevel);
	float imax = ICENTER - (RMIN / zoomLevel);
	float imin = ICENTER + (RMIN / zoomLevel);
	
#endif
#else
	float rmax = RCENTER - (RMIN / exp2f(zoomLevel));
	float rmin = RCENTER + (RMIN / exp2f(zoomLevel));
	float imax = ICENTER - (RMIN / exp2f(zoomLevel));
	float imin = ICENTER + (RMIN / exp2f(zoomLevel));
#endif
	float cr=  x * ((rmax - rmin) / WIDTH) + rmin;
	float ci = y * ((imax - imin) / HEIGHT) + imin;

	//float cr = x * ((RMAX - RMIN) / WIDTH) + RMIN;
	//float ci = y * ((IMAX - IMIN) / HEIGHT) + IMIN;
	//float cr = x * ((md->rmax - md->rmin) / WIDTH) + md->rmin;
	//float ci = y * ((md->imax - md->imin) / HEIGHT) + md->imin;
# if DEBUG_GPU
	int point = y * WIDTH + x;

	if (point == 0) {
#if  SEC_APPROACH
		printf("Thread %i, ,zoom=%f \n", point, zoomLevel);
#else
		printf("Thread %i, ,zoom=%f \n", point, exp2f(zoomLevel));		
#endif
		//printf("Thread %i ,threadIdx.y=%i,block x=%i, block y=%i,width=%i, height=%i, imax=%i ,imin=%f, rmax=%f ,rmin=%f, \n", point, threadIdx.y, blockDim.x, blockDim.y, WIDTH, HEIGHT, (int)md->imax, md->imin, md->rmax, md->rmin);
	}
#endif
	pfc::byte_t  i = 0;
	float zr = 0.0, zi = 0.0;

#if LOOPUNROLLING
	#pragma unroll (NMBLOOPS2UNROLL)
#endif
#if FORLOOP
	for (; i < MAXITER && zr*zr + zi * zi < LIMIT; i++) {
#if PERIODICITY_CHECKING
		float rtemp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		float itemp = 2.0f * zr * zi + ci;
		if (zi == itemp && zr == rtemp) {
			i = MAXITER;
			break;
		}
		zi = itemp;
		zr = rtemp; //now it is okay

#else
		//float zi2 = zi * zi;				 // Instruction Parallelisem 
		float temp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		zi = 2.0f * zr * zi + ci;
		zr = temp; //now it is okay
#endif
	}
#else
	while (zr*zr + zi * zi < LIMIT && i < MAXITER) {
#if PERIODICITY_CHECKING
		float rtemp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		float itemp = 2.0f * zr * zi + ci;
		if (zi == itemp && zr == rtemp) {
			i = MAXITER;
			break;
		}
		zi = itemp;
		zr = rtemp; //now it is okay
		i++

#else
		float temp = zr * zr - zi * zi + cr; // need save the value in temp in order to not overwrite the old value
		zi = 2.0f * zr * zi + ci;
		zr = temp; //now it is okay
		i++;
#endif
	}
#endif
	// compute  pixel color and write it to file 
	bmp[y * WIDTH + x] = {
		static_cast<pfc::byte_t>(((2 * i))),
		static_cast<pfc::byte_t>(((3 * i))),
		static_cast<pfc::byte_t>((3 * i))
	};
}


 void  Fractal_Carpat_GPU(pfc::BGR_4_t * bmp, float zoomLevel) {

	dim3 ThreadsPerBlock(THREADSPERBLOCK, THREADSPERBLOCK);
	dim3 blocksperGrid(WIDTH /ThreadsPerBlock.x, HEIGHT /ThreadsPerBlock.y);
	call_kernel << <blocksperGrid, ThreadsPerBlock >> > (bmp, zoomLevel);

}

 void  Fractal_Carpat_GPU(pix_t * bmp, hipStream_t stream, float zoomLevel) {

  dim3 ThreadsPerBlock(THREADSPERBLOCK, THREADSPERBLOCK);
  dim3 blocksperGrid(WIDTH / ThreadsPerBlock.x, HEIGHT / ThreadsPerBlock.y);
  call_kernel << <blocksperGrid, ThreadsPerBlock, 0, stream >> > (bmp, zoomLevel);

 }


